
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>


#define SIZE 10485760
#define ITERATIONS 65536

__global__
void average(double *invec, double *outvec)
{
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if(idx <= 0 || idx >= SIZE + 1)
                return;
        outvec[idx] = (invec[idx-1] + invec[idx+1]) / 2.0;
}

int main()
{
        // Crate host and device pointers for input and output arrays
        double *invec, *outvec;
        double *d_invec, *d_outvec;

        // Allocate and initialize arrays on host side
        invec = (double*)malloc((SIZE + 2) * sizeof(double));
        outvec = (double*)malloc((SIZE + 2) * sizeof(double));
        memset(invec, 0, sizeof(double) * (SIZE + 2));
        memset(outvec, 0, sizeof(double) * (SIZE + 2));
        invec[SIZE + 1] = 1.0;

        // Allocate and initialize memory on device side
        hipMalloc((void**)&d_invec, (SIZE + 2) * sizeof(double));
        hipMalloc((void**)&d_outvec, (SIZE + 2) * sizeof(double));
        hipMemset(d_outvec, 0, (SIZE + 2) * sizeof(double));
        
        // Copy the data to the device
        hipMemcpy(d_invec, invec, (SIZE + 2) * sizeof(double), hipMemcpyHostToDevice);

        // Create CUDA events to time the kernel execution
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Run iterative averaging
        hipEventRecord(start);
        for(int i = 0; i < ITERATIONS; i++)
        {
                if(i%2) // Odd iterations
                        average<<<8192, (SIZE + 2)/8192>>>(d_outvec, d_invec);
                else    // Even iterations
                        average<<<8192, (SIZE + 2)/8192>>>(d_invec, d_outvec);
        }
        hipEventRecord(stop);

        // Copy the data back to host
        hipMemcpy(outvec, d_outvec, (SIZE + 2) * sizeof(float), hipMemcpyDeviceToHost);
        
        // Compute the time taken for execution
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout  << "Time taken : " << milliseconds << "ms" << std::endl;

        // Free up memory on device side
        hipFree(d_invec);
        hipFree(d_outvec);
        
        // Free up memory on host side
        delete(invec);
        delete(outvec);
}
